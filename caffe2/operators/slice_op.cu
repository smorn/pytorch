#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/slice_op.h"
#include "caffe2/utils/math.h"

namespace caffe2 {
namespace {
__global__ void SliceCopyKernel(
    char* src_offset_bytes,
    int src_frame_size_bytes,
    char* dst_offset_bytes,
    int dst_frame_size_bytes,
    int copy_size) {
  if ((copy_size % sizeof(int) == 0) &&
      (src_frame_size_bytes % sizeof(int) == 0) &&
      (dst_frame_size_bytes % sizeof(int) == 0)) {
    int* src = (int*)src_offset_bytes;
    int* dst = (int*)dst_offset_bytes;

    int src_frame_size = src_frame_size_bytes / sizeof(int);
    int dst_frame_size = dst_frame_size_bytes / sizeof(int);

    int copyChunks = copy_size / sizeof(int);

    CUDA_1D_KERNEL_LOOP(index, copyChunks) {
      int chunk = index % copyChunks;
      int block = index / copyChunks;

      dst[block * dst_frame_size + chunk] = src[block * src_frame_size + chunk];
    }
  } else {
    char* src = (char*)src_offset_bytes;
    char* dst = (char*)dst_offset_bytes;

    int src_frame_size = src_frame_size_bytes / sizeof(char);
    int dst_frame_size = dst_frame_size_bytes / sizeof(char);

    int copyChunks = copy_size / sizeof(char);

    CUDA_1D_KERNEL_LOOP(index, copyChunks) {
      int chunk = index % copyChunks;
      int block = index / copyChunks;

      dst[block * dst_frame_size + chunk] = src[block * src_frame_size + chunk];
    }
  }
}

template <class SIndex, class Context>
bool SliceImplGpu(
    Tensor<Context>* output,
    const Tensor<Context>& data,
    const TensorCPU& starts,
    const TensorCPU& ends,
    Context* context,
    Tensor<Context>* gdata = nullptr,
    const Tensor<Context>* go = nullptr) {
  bool backward = output == nullptr;

  auto* starts_data = starts.template data<SIndex>();
  auto* ends_data = ends.template data<SIndex>();

  CAFFE_ENFORCE_EQ(starts.ndim(), 1);
  CAFFE_ENFORCE_EQ(ends.ndim(), 1);
  CAFFE_ENFORCE_GE(data.ndim(), starts.size());
  CAFFE_ENFORCE_EQ(starts.size(), ends.size());

  std::vector<int> starts_idx(data.ndim());
  std::vector<int> ends_idx(data.ndim());
  std::vector<int> dst_sizes(data.ndim());

  for (int i = 0; i < data.ndim(); ++i) {
    if (i >= starts.size()) {
      starts_idx[i] = 0;
      ends_idx[i] = data.dims()[i];
      continue;
    }
    if (data.dims()[i] > 0) {
      auto start = starts_data[i];
      auto end = ends_data[i];
      if (start < 0) {
        start = data.dims()[i] + 1 + start;
      }
      if (end < 0) {
        end = data.dims()[i] + 1 + end;
      }
      CAFFE_ENFORCE_GE(start, 0);
      CAFFE_ENFORCE_GE(end, 0);
      CAFFE_ENFORCE_LT(start, data.dims()[i]);
      CAFFE_ENFORCE_LE(end, data.dims()[i]);
      CAFFE_ENFORCE_GE(end, start);
      starts_idx[i] = start;
      ends_idx[i] = end;
      dst_sizes[i] = end - start;
    } else {
      starts_idx[i] = 0;
      ends_idx[i] = 0;
      dst_sizes[i] = 0;
    }
  }

  if (data.size() <= 0) {
    // When the input is empty, we do not need to do copy.
    if (!backward) {
      output->Resize(dst_sizes);
      output->raw_mutable_data(data.meta());
    }
    return true;
  }
  // for now only supports slicing in 1 dimension
  int dim = -1;
  for (int i = 0; i < data.ndim(); ++i) {
    if (starts_idx[i] > 0 || ends_idx[i] < data.dims()[i])
      dim = i;
  }
  if (dim == -1) {
    if (!backward) {
      output->CopyFrom(data, context);
    } else {
      gdata->CopyFrom(*go, context);
    }
    return true;
  }
  int unit = std::accumulate(
      data.dims().begin() + dim + 1,
      data.dims().end(),
      1,
      std::multiplies<int>());
  int num_blocks = std::accumulate(
      data.dims().begin(),
      data.dims().begin() + dim,
      1,
      std::multiplies<int>());
  if (!backward) {
    output->Resize(dst_sizes);
  } else {
    gdata->ResizeLike(data);
  }

  auto itemsize = data.meta().itemsize();

  if (!backward) {
    char* src_bytes = (char*)data.raw_data();
    char* dst_bytes = (char*)output->raw_mutable_data(data.meta());

    size_t src_nbytes = data.nbytes();
    size_t dst_nbytes = output->nbytes();

    size_t src_frame_size = unit * data.dims()[dim];
    size_t dst_frame_size = unit * (ends_idx[dim] - starts_idx[dim]);
    size_t src_offset = unit * starts_idx[dim];

    if (num_blocks == 0 || dst_frame_size == 0) {
      return true;
    }

    size_t src_frame_size_bytes = itemsize * src_frame_size;
    size_t dst_frame_size_bytes = itemsize * dst_frame_size;
	
	std::vector<int> it(dim);
	for(int i = 0 ; i < dim ; i++) it[i] = starts_idx[i];
	while(it[0] < ends_idx[0]) {
		//get the start postion of current frame
		size_t src_frame_idx = 0;
		size_t dst_frame_idx = 0;
		size_t src_basis = 1;
		size_t dst_basis = 1;
		for(int d = dim - 1 ; d >= 0 ; d--) {
			src_frame_idx += it[d] * src_basis;
			dst_frame_idx += (it[d] - starts_idx[d]) * dst_basis;
			src_basis *= data.dims()[d];
			dst_basis *= dst_sizes[d];
		}
		//copy the block in current src frame to dst frame
		char * local_src_offset_bytes = src_bytes + src_frame_idx * src_frame_size_bytes + itemsize * src_offset;
		char * local_dst_offset_bytes = dst_bytes + dst_frame_idx * dst_frame_size_bytes;
		SliceCopyKernel<<<
			std::min(num_blocks, CAFFE_MAXIMUM_NUM_BLOCKS),
			CAFFE_CUDA_NUM_THREADS,
			0,
			context->hip_stream()>>>(
				local_src_offset_bytes,
				src_frame_size_bytes,
				local_dst_offset_bytes,
				dst_frame_size_bytes,
				dst_frame_size_bytes);
		//get the index of next frame
		it[dim - 1]++;
		for(int i = dim - 1 ; i > 0 && (it[i] >= ends_idx[i]) ; --i) {
			it[i] = starts_idx[i];
			it[i - 1]++;
		}
	}
  } else {
    char* src_bytes = (char*)go->raw_data();
    char* dst_bytes = (char*)gdata->raw_mutable_data(go->meta());

    size_t src_nbytes = go->nbytes();
    size_t dst_nbytes = gdata->nbytes();

    size_t src_frame_size = unit * (ends_idx[dim] - starts_idx[dim]);
    size_t dst_frame_size = unit * data.dims()[dim];
    size_t dst_offset = unit * starts_idx[dim];

    if (num_blocks == 0 || dst_frame_size == 0) {
      return true;
    }

    size_t src_frame_size_bytes = itemsize * src_frame_size;
    size_t dst_frame_size_bytes = itemsize * dst_frame_size;

    // Zero out gradient blob before copy since we copy in fewer items than
    // there is space for
    math::Set<float, HIPContext>(
        gdata->size(),
        0.0f,
        (float*)gdata->raw_mutable_data(go->meta()),
        context);

    // If output tensor is empty, just return zeroed gradient tensor
    if (!src_bytes) {
      return true;
    }

    std::vector<int> it(dim);
	for(int i = 0 ; i < dim ; i++) it[i] = starts_idx[i];
	while(it[0] < ends_idx[0]) {
		//get the start postion of current frame
		size_t src_frame_idx = 0;
		size_t dst_frame_idx = 0;
		size_t src_basis = 1;
		size_t dst_basis = 1;
		for(int d = dim - 1 ; d >= 0 ; d--) {
			src_frame_idx += (it[d] - starts_idx[d]) * src_basis;
			dst_frame_idx += it[d] * dst_basis;
			src_basis *= dst_sizes[d];
			dst_basis *= data.dims()[d];
		}
		//copy current srd frame to the block in dst frame
		char * local_src_offset_bytes = src_bytes + src_frame_idx * src_frame_size_bytes;
		char * local_dst_offset_bytes = dst_bytes + dst_frame_idx * dst_frame_size_bytes + itemsize * dst_offset;
		SliceCopyKernel<<<
			std::min(num_blocks, CAFFE_MAXIMUM_NUM_BLOCKS),
			CAFFE_CUDA_NUM_THREADS,
			0,
			context->hip_stream()>>>(
				local_src_offset_bytes,
				src_frame_size_bytes,
				local_dst_offset_bytes,
				dst_frame_size_bytes,
				src_frame_size_bytes);
		//get the index of next frame
		it[dim - 1]++;
		for(int i = dim - 1 ; i > 0 && (it[i] >= ends_idx[i]) ; --i) {
			it[i] = starts_idx[i];
			it[i - 1]++;
		}		
	}
  }

  return true;
}

} // namespace

template <>
bool SliceOp<int, HIPContext>::RunOnDevice() {
  auto* output = Output(0);
  auto& data = Input(0);

  if (InputSize() > 1) {
    starts_host_.CopyFrom<HIPContext>(Input(1));
    ends_host_.CopyFrom<HIPContext>(Input(2));
  } else {
    if (!statically_inited_) {
      CAFFE_ENFORCE(HasArgument("starts"));
      CAFFE_ENFORCE(HasArgument("ends"));
      CAFFE_ENFORCE_EQ(starts_.size(), ends_.size());

      starts_host_.Resize(starts_.size());
      ends_host_.Resize(ends_.size());

      memcpy(
          starts_host_.mutable_data<int>(),
          starts_.data(),
          sizeof(int) * starts_.size());
      memcpy(
          ends_host_.mutable_data<int>(),
          ends_.data(),
          sizeof(int) * ends_.size());
      statically_inited_ = true;
    }
  }

  return SliceImplGpu<int, HIPContext>(
      output, data, starts_host_, ends_host_, &context_);
}

REGISTER_HIP_OPERATOR(Slice, SliceOp<int, HIPContext>);

template <>
bool SliceGradientOp<int, HIPContext>::RunOnDevice() {
  auto* gdata = Output(0);
  auto& data = Input(0);

  if (InputSize() == 4) {
    starts_host_.CopyFrom<HIPContext>(Input(1));
    ends_host_.CopyFrom<HIPContext>(Input(2));

    auto& go = Input(3);

    return SliceImplGpu<int, HIPContext>(
        nullptr, data, starts_host_, ends_host_, &context_, gdata, &go);
  } else {
    if (!statically_inited_) {
      CAFFE_ENFORCE(HasArgument("starts"));
      CAFFE_ENFORCE(HasArgument("ends"));
      CAFFE_ENFORCE_EQ(starts_.size(), ends_.size());

      starts_host_.Resize(starts_.size());
      ends_host_.Resize(ends_.size());

      memcpy(
          starts_host_.mutable_data<int>(),
          starts_.data(),
          sizeof(int) * starts_.size());
      memcpy(
          ends_host_.mutable_data<int>(),
          ends_.data(),
          sizeof(int) * ends_.size());

      statically_inited_ = true;
    }
    auto& go = Input(1);

    return SliceImplGpu<int, HIPContext>(
        nullptr, data, starts_host_, ends_host_, &context_, gdata, &go);
  }
}
REGISTER_HIP_OPERATOR(SliceGradient, SliceGradientOp<int, HIPContext>);
} // namespace caffe2
